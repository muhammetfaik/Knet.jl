#include "hip/hip_runtime.h"
#include "../knet.h"

template<typename dType>
__global__ void _add_csr_dns_atomic(int m, int n, dType alpha,
			     int nnzA,
			     const dType *csrValA,
			     const int *csrRowPtrA,
			     const int *csrColIndA,
			     dType *B) {
  int nz = threadIdx.x + blockIdx.x * blockDim.x;
  int row = 0;
  while (nz < nnzA) {
    dType val = alpha * csrValA[nz];
    int col = csrColIndA[nz]-1;
    for (; nz > csrRowPtrA[row+1]-2; row++);
    atomicAdd(&B[col * m + row], val);
    nz += blockDim.x * gridDim.x;
  }
}

extern "C" {
  void add_csr_dns_atomic_32(int m, int n, float  alpha, int nnzA, const float  *csrValA, const int *csrRowPtrA, const int *csrColIndA, float  *B) KCALL(_add_csr_dns_atomic,m,n,alpha,nnzA,csrValA,csrRowPtrA,csrColIndA,B);
  void add_csr_dns_atomic_64(int m, int n, double alpha, int nnzA, const double *csrValA, const int *csrRowPtrA, const int *csrColIndA, double *B) KCALL(_add_csr_dns_atomic,m,n,alpha,nnzA,csrValA,csrRowPtrA,csrColIndA,B);
}

template<typename dType>
__global__ void _add_csr_dns(int m, int n, dType alpha,
			     int nnzA,
			     const dType *csrValA,
			     const int *csrRowPtrA,
			     const int *csrColIndA,
			     dType *B) {
  int nz = threadIdx.x + blockIdx.x * blockDim.x;
  int row = 0;
  while (nz < nnzA) {
    dType val = alpha * csrValA[nz];
    int col = csrColIndA[nz]-1;
    for (; nz > csrRowPtrA[row+1]-2; row++);
    B[col * m + row] +=  val;
    nz += blockDim.x * gridDim.x;
  }
}

extern "C" {
  void add_csr_dns_32(int m, int n, float  alpha, int nnzA, const float  *csrValA, const int *csrRowPtrA, const int *csrColIndA, float  *B) KCALL(_add_csr_dns,m,n,alpha,nnzA,csrValA,csrRowPtrA,csrColIndA,B);
  void add_csr_dns_64(int m, int n, double alpha, int nnzA, const double *csrValA, const int *csrRowPtrA, const int *csrColIndA, double *B) KCALL(_add_csr_dns,m,n,alpha,nnzA,csrValA,csrRowPtrA,csrColIndA,B);
}

template<typename dType>
__global__ void _add_csc_dns_atomic(int m, int n, dType alpha,
				    int nnzA,
				    const dType *cscValA,
				    const int *cscColPtrA,
				    const int *cscRowIndA,
				    dType *B) {
  int nz = threadIdx.x + blockIdx.x * blockDim.x;
  int col = 0;
  while (nz < nnzA) {
    dType val = alpha * cscValA[nz];
    int row = cscRowIndA[nz]-1;
    for (; nz > cscColPtrA[col+1]-2; col++);
    atomicAdd(&B[col * m + row], val);
    nz += blockDim.x * gridDim.x;
  }
}

extern "C" {
  void add_csc_dns_atomic_32(int m, int n, float  alpha, int nnzA, const float  *cscValA, const int *cscColPtrA, const int *cscRowIndA, float  *B) KCALL(_add_csc_dns_atomic,m,n,alpha,nnzA,cscValA,cscColPtrA,cscRowIndA,B);
  void add_csc_dns_atomic_64(int m, int n, double alpha, int nnzA, const double *cscValA, const int *cscColPtrA, const int *cscRowIndA, double *B) KCALL(_add_csc_dns_atomic,m,n,alpha,nnzA,cscValA,cscColPtrA,cscRowIndA,B);
}

template<typename dType>
__global__ void _add_csc_dns(int m, int n, dType alpha,
			     int nnzA,
			     const dType *cscValA,
			     const int *cscColPtrA,
			     const int *cscRowIndA,
			     dType *B) {
  int nz = threadIdx.x + blockIdx.x * blockDim.x;
  int col = 0;
  while (nz < nnzA) {
    dType val = alpha * cscValA[nz];
    int row = cscRowIndA[nz]-1;
    for (; nz > cscColPtrA[col+1]-2; col++);
    B[col * m + row] +=  val;
    nz += blockDim.x * gridDim.x;
  }
}

extern "C" {
  void add_csc_dns_32(int m, int n, float  alpha, int nnzA, const float  *cscValA, const int *cscColPtrA, const int *cscRowIndA, float  *B) KCALL(_add_csc_dns,m,n,alpha,nnzA,cscValA,cscColPtrA,cscRowIndA,B);
  void add_csc_dns_64(int m, int n, double alpha, int nnzA, const double *cscValA, const int *cscColPtrA, const int *cscRowIndA, double *B) KCALL(_add_csc_dns,m,n,alpha,nnzA,cscValA,cscColPtrA,cscRowIndA,B);
}

/*
   ........x............................................
   ..................x..................................
   ..x..................................................
...
...
...
...
*/

/* dw=dy*xt where dy is dense column major, xt is sparse csr (or
   transposed csc) and dw is sparse csr.  each row of dw will hold
   nnz(x) values.  crows assumed initialized.  resulting ccols 
   unsorted with possible duplicates. */

template<typename dType>
__global__ void _mul_dns_csr_csru(int arows, int acols, dType *a, int *brows, int *bcols, dType *b, int *crows, int *ccols, dType *c) {
  int t = threadIdx.x + blockIdx.x * blockDim.x;
  int T = arows*acols;
  int bnnz = brows[acols]-1;
  if (t == 0) crows[0] = 1;
  while (t < T) {
    dType aval = a[t];
    int arow0 = t % arows;
    int acol0 = t / arows;
    int brow0 = acol0;
    int crow0 = arow0;
    for (int b1 = brows[brow0]; b1 < brows[brow0+1]; b1++) { 	// b[brows[r]-1...brows[r+1]-2] contains row 0-based-r
      int b0 = b1-1;
      int c0 = crow0 * bnnz + b0;				// each c row has nnz(b) entries so we can use nz index of b as row index of c
      ccols[c0] = bcols[b0];
      c[c0] = aval * b[b0];
    }
    if (acol0 == 0) crows[crow0+1] = (crow0+1)*bnnz + 1;
    t += blockDim.x * gridDim.x;
  }
}

extern "C" {
  void mul_dns_csr_csru_32(int arows, int acols, float  *a, int *brows, int *bcols, float  *b, int *crows, int *ccols, float  *c) KCALL(_mul_dns_csr_csru, arows, acols, a, brows, bcols, b, crows, ccols, c);
  void mul_dns_csr_csru_64(int arows, int acols, double *a, int *brows, int *bcols, double *b, int *crows, int *ccols, double *c) KCALL(_mul_dns_csr_csru, arows, acols, a, brows, bcols, b, crows, ccols, c);
}


template<typename dType>
__global__ void _mul_csc_dns_cscu(int brows, int bcols, int *acols, int *arows, dType *a, dType *b, int *ccols, int *crows, dType *c) {
  int t = threadIdx.x + blockIdx.x * blockDim.x;
  int T = brows*bcols;
  int annz = acols[brows]-1;
  if (t == 0) ccols[0] = 1;
  while (t < T) {
    dType bval = b[t];
    int brow0 = t % brows;
    int bcol0 = t / brows;
    int acol0 = brow0;
    int ccol0 = bcol0;
    for (int a1 = acols[acol0]; a1 < acols[acol0+1]; a1++) { 	// a[acols[r]-1...acols[r+1]-2] contains col 0-based-r
      int a0 = a1-1;
      int c0 = ccol0 * annz + a0;				// each c col has nnz(a) entries so we can use nz index of a as col index of c
      crows[c0] = arows[a0];
      c[c0] = bval * a[a0];
    }
    if (brow0 == 0) ccols[ccol0+1] = (ccol0+1)*annz + 1;
    t += blockDim.x * gridDim.x;
  }
}

extern "C" {
  void mul_csc_dns_cscu_32(int brows, int bcols, int *acols, int *arows, float  *a, float  *b, int *ccols, int *crows, float  *c) KCALL(_mul_csc_dns_cscu,brows,bcols,acols,arows,a,b,ccols,crows,c);
  void mul_csc_dns_cscu_64(int brows, int bcols, int *acols, int *arows, double *a, double *b, int *ccols, int *crows, double *c) KCALL(_mul_csc_dns_cscu,brows,bcols,acols,arows,a,b,ccols,crows,c);
}


/* CUBLAS nrm2 is extremely slow.  The following is a substitute from Barret Zoph.
   Based on: http://developer.download.nvidia.com/compute/cuda/1.1-Beta/x86_website/projects/reduction/doc/reduction.pdf
*/

//for optimizing warps
//volatile must be used as register optimization will lead to wrong answers
template<typename dType>
__device__ 
void warpReduceSum(volatile dType* sdata, int tid) {
	sdata[tid] += sdata[tid + 32];
	sdata[tid] += sdata[tid + 16];
	sdata[tid] += sdata[tid + 8];
	sdata[tid] += sdata[tid + 4];
	sdata[tid] += sdata[tid + 2];
	sdata[tid] += sdata[tid + 1];
}

#define NORM_THREADS 256
template<typename dType>
__global__
void basic_compute_norm_p1(dType *d_gradient,int size,dType *result) {
	__shared__ dType buffer[NORM_THREADS];
	int i_start = threadIdx.x+blockIdx.x*blockDim.x; //start at the thread index
	int i_end = size; //end at dim
	int i_step = blockDim.x*gridDim.x; //the block dimension (aka the number of threads in the block) is the step
	int tid = threadIdx.x;


	buffer[tid] = 0;
	for(int i= i_start; i<i_end; i+=i_step) {
		buffer[tid]+=(d_gradient[i]*d_gradient[i]);
	}
	__syncthreads();

	for(int stride=NORM_THREADS/2; stride>32; stride>>=1) {
		if(tid < stride) {
			buffer[tid] += buffer[stride + tid];
		}
		__syncthreads();
	}

	if(tid<32) {
		warpReduceSum(buffer,tid);
	}
	__syncthreads();

	if(tid==0) {
		result[blockIdx.x]=buffer[0];
	}
}


template<typename dType>
__global__
void basic_compute_norm_p2(dType *temp_result,dType *final_result) {
	__shared__ dType buffer[NORM_THREADS];

	int tid = threadIdx.x;
	buffer[tid] = temp_result[tid];
	__syncthreads();

	for(int stride=NORM_THREADS/2; stride>32; stride>>=1) {
		if(tid < stride) {
			buffer[tid] += buffer[stride + tid];
		}
		__syncthreads();
	}

	if(tid<32) {
		warpReduceSum(buffer,tid);
	}
	__syncthreads();

	if(tid==0) {
		final_result[0]=buffer[0];
	}
}

template<typename dType>
__global__
void basic_compute_asum_p1(dType *d_gradient,int size,dType *result) {
	__shared__ dType buffer[NORM_THREADS];
	int i_start = threadIdx.x+blockIdx.x*blockDim.x; //start at the thread index
	int i_end = size; //end at dim
	int i_step = blockDim.x*gridDim.x; //the block dimension (aka the number of threads in the block) is the step
	int tid = threadIdx.x;


	buffer[tid] = 0;
	for(int i= i_start; i<i_end; i+=i_step) {
	  dType di = d_gradient[i];
	  if (di < 0) di = -di;
	  buffer[tid]+=di;
	}
	__syncthreads();

	for(int stride=NORM_THREADS/2; stride>32; stride>>=1) {
		if(tid < stride) {
			buffer[tid] += buffer[stride + tid];
		}
		__syncthreads();
	}

	if(tid<32) {
		warpReduceSum(buffer,tid);
	}
	__syncthreads();

	if(tid==0) {
		result[blockIdx.x]=buffer[0];
	}
}


template<typename dType>
__global__
void basic_compute_asum_p2(dType *temp_result,dType *final_result) {
	__shared__ dType buffer[NORM_THREADS];

	int tid = threadIdx.x;
	buffer[tid] = temp_result[tid];
	__syncthreads();

	for(int stride=NORM_THREADS/2; stride>32; stride>>=1) {
		if(tid < stride) {
			buffer[tid] += buffer[stride + tid];
		}
		__syncthreads();
	}

	if(tid<32) {
		warpReduceSum(buffer,tid);
	}
	__syncthreads();

	if(tid==0) {
		final_result[0]=buffer[0];
	}
}

template<typename dType>
__global__
void basic_compute_sum_p1(dType *d_gradient,int size,dType *result) {
	__shared__ dType buffer[NORM_THREADS];
	int i_start = threadIdx.x+blockIdx.x*blockDim.x; //start at the thread index
	int i_end = size; //end at dim
	int i_step = blockDim.x*gridDim.x; //the block dimension (aka the number of threads in the block) is the step
	int tid = threadIdx.x;


	buffer[tid] = 0;
	for(int i= i_start; i<i_end; i+=i_step) {
	  buffer[tid]+=d_gradient[i];
	}
	__syncthreads();

	for(int stride=NORM_THREADS/2; stride>32; stride>>=1) {
		if(tid < stride) {
			buffer[tid] += buffer[stride + tid];
		}
		__syncthreads();
	}

	if(tid<32) {
		warpReduceSum(buffer,tid);
	}
	__syncthreads();

	if(tid==0) {
		result[blockIdx.x]=buffer[0];
	}
}


template<typename dType>
__global__
void basic_compute_sum_p2(dType *temp_result,dType *final_result) {
	__shared__ dType buffer[NORM_THREADS];

	int tid = threadIdx.x;
	buffer[tid] = temp_result[tid];
	__syncthreads();

	for(int stride=NORM_THREADS/2; stride>32; stride>>=1) {
		if(tid < stride) {
			buffer[tid] += buffer[stride + tid];
		}
		__syncthreads();
	}

	if(tid<32) {
		warpReduceSum(buffer,tid);
	}
	__syncthreads();

	if(tid==0) {
		final_result[0]=buffer[0];
	}
}


extern "C" {
  float vecnorm2_32(float *d_array,int size) {
    float norm;
    static float *d_temp_result;
    static float *d_result;
    if (d_temp_result == NULL) hipMalloc(&d_temp_result, NORM_THREADS*sizeof(float));
    if (d_result == NULL) hipMalloc(&d_result, 1*sizeof(float));
    basic_compute_norm_p1<<<NORM_THREADS,NORM_THREADS>>>(d_array,size,d_temp_result);
    basic_compute_norm_p2<<<1,NORM_THREADS>>>(d_temp_result,d_result);
    hipMemcpy(&norm,d_result,1*sizeof(float),hipMemcpyDeviceToHost);
    return sqrt(norm);
  }

  double vecnorm2_64(double *d_array,int size) {
    double norm;
    static double *d_temp_result;
    static double *d_result;
    if (d_temp_result == NULL) hipMalloc(&d_temp_result, NORM_THREADS*sizeof(double));
    if (d_result == NULL) hipMalloc(&d_result, 1*sizeof(double));
    basic_compute_norm_p1<<<NORM_THREADS,NORM_THREADS>>>(d_array,size,d_temp_result);
    basic_compute_norm_p2<<<1,NORM_THREADS>>>(d_temp_result,d_result);
    hipMemcpy(&norm,d_result,1*sizeof(double),hipMemcpyDeviceToHost);
    return sqrt(norm);
  }

  // Unfortunately vecnorm1 is not any faster than CUBLAS.asum

  float vecnorm1_32(float *d_array,int size) {
    float norm;
    static float *d_temp_result;
    static float *d_result;
    if (d_temp_result == NULL) hipMalloc(&d_temp_result, NORM_THREADS*sizeof(float));
    if (d_result == NULL) hipMalloc(&d_result, 1*sizeof(float));
    basic_compute_asum_p1<<<NORM_THREADS,NORM_THREADS>>>(d_array,size,d_temp_result);
    basic_compute_asum_p2<<<1,NORM_THREADS>>>(d_temp_result,d_result);
    hipMemcpy(&norm,d_result,1*sizeof(float),hipMemcpyDeviceToHost);
    return norm;
  }

  double vecnorm1_64(double *d_array,int size) {
    double norm;
    static double *d_temp_result;
    static double *d_result;
    if (d_temp_result == NULL) hipMalloc(&d_temp_result, NORM_THREADS*sizeof(double));
    if (d_result == NULL) hipMalloc(&d_result, 1*sizeof(double));
    basic_compute_asum_p1<<<NORM_THREADS,NORM_THREADS>>>(d_array,size,d_temp_result);
    basic_compute_asum_p2<<<1,NORM_THREADS>>>(d_temp_result,d_result);
    hipMemcpy(&norm,d_result,1*sizeof(double),hipMemcpyDeviceToHost);
    return norm;
  }

  // Here is regular sum
  float sum32(float *d_array,int size) {
    float norm;
    static float *d_temp_result;
    static float *d_result;
    if (d_temp_result == NULL) hipMalloc(&d_temp_result, NORM_THREADS*sizeof(float));
    if (d_result == NULL) hipMalloc(&d_result, 1*sizeof(float));
    basic_compute_sum_p1<<<NORM_THREADS,NORM_THREADS>>>(d_array,size,d_temp_result);
    basic_compute_sum_p2<<<1,NORM_THREADS>>>(d_temp_result,d_result);
    hipMemcpy(&norm,d_result,1*sizeof(float),hipMemcpyDeviceToHost);
    return norm;
  }

  double sum64(double *d_array,int size) {
    double norm;
    static double *d_temp_result;
    static double *d_result;
    if (d_temp_result == NULL) hipMalloc(&d_temp_result, NORM_THREADS*sizeof(double));
    if (d_result == NULL) hipMalloc(&d_result, 1*sizeof(double));
    basic_compute_sum_p1<<<NORM_THREADS,NORM_THREADS>>>(d_array,size,d_temp_result);
    basic_compute_sum_p2<<<1,NORM_THREADS>>>(d_temp_result,d_result);
    hipMemcpy(&norm,d_result,1*sizeof(double),hipMemcpyDeviceToHost);
    return norm;
  }
}

/*
  The following functions multiply two sparse matrices into a dense matrix.
  The sparse matrices are in 1-based csc format.
  Ast_mul_Bs uses the transpose of the first arg and a simpler algorithm.
  x(nd,nx) s(nd,ns) -> k(nx,ns)
  As_mul_Bs uses the fast algorithm from the Julia sparse code.
  x(nx,nd) s(nd,ns) -> k(nx,ns)
  The difference in speed is significant on the CPU but only around 50% on the GPU
*/

__global__ void _Ast_mul_Bs_32(int nx, int ns, float *xval, int *xrow, int *xcol, float *sval, int *srow, int *scol, float *k) {
  int i, n, x1, x2, xc, xr, s1, s2, sc, sr;
  i = threadIdx.x + blockIdx.x * blockDim.x;
  n = nx*ns;
  while (i < n) {
    double ki = 0;
    xc = i % nx;
    sc = i / nx;
    x1 = xcol[xc]-1; x2 = xcol[xc+1]-1;
    s1 = scol[sc]-1; s2 = scol[sc+1]-1;
    while ((x1 < x2) && (s1 < s2)) {
      xr = xrow[x1]; sr = srow[s1];
      if (sr < xr) s1++;
      else if (xr < sr) x1++;
      else ki += xval[x1++]*sval[s1++];
    }
    k[i] = ki;
    i += blockDim.x * gridDim.x;
  }
}

__global__ void _Ast_mul_Bs_64(int nx, int ns, double *xval, int *xrow, int *xcol, double *sval, int *srow, int *scol, double *k) {
  int i, n, x1, x2, xc, xr, s1, s2, sc, sr;
  i = threadIdx.x + blockIdx.x * blockDim.x;
  n = nx*ns;
  while (i < n) {
    double ki = 0;
    xc = i % nx;
    sc = i / nx;
    x1 = xcol[xc]-1; x2 = xcol[xc+1]-1;
    s1 = scol[sc]-1; s2 = scol[sc+1]-1;
    while ((x1 < x2) && (s1 < s2)) {
      xr = xrow[x1]; sr = srow[s1];
      if (sr < xr) s1++;
      else if (xr < sr) x1++;
      else ki += xval[x1++]*sval[s1++];
    }
    k[i] = ki;
    i += blockDim.x * gridDim.x;
  }
}

__global__ void _As_mul_Bs_32(int mx, int ns, float *xval, int *xrow, int *xcol, float *sval, int *srow, int *scol, float *k) {
  int s0, s1, sp, sc, sr, x0, x1, xp, xc, xr, k0, k1, kp;
  float sv, xv;
  sc = threadIdx.x + blockIdx.x * blockDim.x;
  while (sc < ns) {	// sc: 0-based column for s
    k0 = mx*sc;		// k[k0]: first element of k[:,sc]
    k1 = k0+mx;		// k[k1-1]: last element of k[:,sc]
    for (kp = k0; kp < k1; kp++) k[kp] = 0;
    s0 = scol[sc]-1;    // first element of s[:,sc] is at sval[s0] (scol entries are 1-based)
    s1 = scol[sc+1]-1;  // last element of s[:,sc] is at sval[s1-1]
    for (sp = s0; sp < s1; sp++) {
      sr = srow[sp]-1;  // sr: 0-based row for s (srow entries are 1-based)
      sv = sval[sp];	// sv: s[sr,sc] (0-based)
      xc = sr;		// xc: 0-based column for x (=sr)
      x0 = xcol[xc]-1;  // first element of x[:,xc] is at xval[x0]
      x1 = xcol[xc+1]-1; // last element of x[:,xc] is at xval[x1-1]
      for (xp = x0; xp < x1; xp++) {
	xr = xrow[xp]-1; // xr: 0-based row for x
	xv = xval[xp];	 // xv: x[xr,xc=sr], now we can set k[xr,sc]
	k[k0+xr] += xv*sv;
      }
    }
    sc += blockDim.x * gridDim.x;
  }
}

__global__ void _As_mul_Bs_64(int mx, int ns, double *xval, int *xrow, int *xcol, double *sval, int *srow, int *scol, double *k) {
  int s0, s1, sp, sc, sr, x0, x1, xp, xc, xr, k0, k1, kp;
  double sv, xv;
  sc = threadIdx.x + blockIdx.x * blockDim.x;
  while (sc < ns) {	// sc: 0-based column for s
    k0 = mx*sc;		// k[k0]: first element of k[:,sc]
    k1 = k0+mx;		// k[k1-1]: last element of k[:,sc]
    for (kp = k0; kp < k1; kp++) k[kp] = 0;
    s0 = scol[sc]-1;    // first element of s[:,sc] is at sval[s0] (scol entries are 1-based)
    s1 = scol[sc+1]-1;  // last element of s[:,sc] is at sval[s1-1]
    for (sp = s0; sp < s1; sp++) {
      sr = srow[sp]-1;  // sr: 0-based row for s (srow entries are 1-based)
      sv = sval[sp];	// sv: s[sr,sc] (0-based)
      xc = sr;		// xc: 0-based column for x (=sr)
      x0 = xcol[xc]-1;  // first element of x[:,xc] is at xval[x0]
      x1 = xcol[xc+1]-1; // last element of x[:,xc] is at xval[x1-1]
      for (xp = x0; xp < x1; xp++) {
	xr = xrow[xp]-1; // xr: 0-based row for x
	xv = xval[xp];	 // xv: x[xr,xc=sr], now we can set k[xr,sc]
	k[k0+xr] += xv*sv;
      }
    }
    sc += blockDim.x * gridDim.x;
  }
}

__global__ void _A_mul_Bs_32(int mx, int ns, float *x, float *sval, int *srow, int *scol, float *k) {
  int s0, s1, sp, sc, sr, x0, xr, k0, k1, kp;
  float sv, xv;
  sc = threadIdx.x + blockIdx.x * blockDim.x;
  while (sc < ns) {	// sc: 0-based column for s and k to be processed
    k0 = mx*sc;		// k[k0]: first element of k[:,sc]
    k1 = k0+mx;		// k[k1-1]: last element of k[:,sc]
    for (kp = k0; kp < k1; kp++) k[kp] = 0;
    s0 = scol[sc]-1;    // first element of s[:,sc] is at sval[s0] (scol entries are 1-based)
    s1 = scol[sc+1]-1;  // last element of s[:,sc] is at sval[s1-1]
    for (sp = s0; sp < s1; sp++) {
      sr = srow[sp]-1;  // sr: 0-based row for s (srow entries are 1-based)
      sv = sval[sp];	// sv: s[sr,sc] (0-based), this value multiplies the sr'th column of x
      x0 = mx*sr;	// x[x0]: first element of x[:,sr]
      for (xr = 0; xr < mx; xr++) {
	xv = x[x0+xr];     // xv: x[xr,sr], now we can set k[xr,sc]
	k[k0+xr] += xv*sv;
      }
    }
    sc += blockDim.x * gridDim.x;
  }
}

__global__ void _A_mul_Bs_64(int mx, int ns, double *x, double *sval, int *srow, int *scol, double *k) {
  int s0, s1, sp, sc, sr, x0, xr, k0, k1, kp;
  double sv, xv;
  sc = threadIdx.x + blockIdx.x * blockDim.x;
  while (sc < ns) {	// sc: 0-based column for s and k to be processed
    k0 = mx*sc;		// k[k0]: first element of k[:,sc]
    k1 = k0+mx;		// k[k1-1]: last element of k[:,sc]
    for (kp = k0; kp < k1; kp++) k[kp] = 0;
    s0 = scol[sc]-1;    // first element of s[:,sc] is at sval[s0] (scol entries are 1-based)
    s1 = scol[sc+1]-1;  // last element of s[:,sc] is at sval[s1-1]
    for (sp = s0; sp < s1; sp++) {
      sr = srow[sp]-1;  // sr: 0-based row for s (srow entries are 1-based)
      sv = sval[sp];	// sv: s[sr,sc] (0-based), this value multiplies the sr'th column of x
      x0 = mx*sr;	// x[x0]: first element of x[:,sr]
      for (xr = 0; xr < mx; xr++) {
	xv = x[x0+xr];     // xv: x[xr,sr], now we can set k[xr,sc]
	k[k0+xr] += xv*sv;
      }
    }
    sc += blockDim.x * gridDim.x;
  }
}


/* We will do dw=dy*x' where x is a sparse matrix one column of x at a time. */

__global__ void _A_mul_Bst_32(int my, int xc, float *dy, float *xval, int *xrow, int *xcol, float *dw) {
  // dw[wr,wc] += dy[yr,yc] * x[xr,xc]  where wr=yr, wc=xr, yc=xc
  int t, n, xp, xr, yp, yr, wp;
  t = threadIdx.x + blockIdx.x * blockDim.x;
  n = xcol[xc+1] - xcol[xc];
  while (t < n) {
    xp = xcol[xc] + t - 1;
    xr = xrow[xp] - 1;
    for (yr = 0; yr < my; yr++) {
      yp = yr + xc * my;
      wp = yr + xr * my;
      dw[wp] += dy[yp] * xval[xp];
    }
    t += blockDim.x * gridDim.x;
  }
}

__global__ void _A_mul_Bst_64(int my, int xc, double *dy, double *xval, int *xrow, int *xcol, double *dw) {
  // dw[wr,wc] += dy[yr,yc] * x[xr,xc]  where wr=yr, wc=xr, yc=xc
  int t, n, xp, xr, yp, yr, wp;
  t = threadIdx.x + blockIdx.x * blockDim.x;
  n = xcol[xc+1] - xcol[xc];
  while (t < n) {
    xp = xcol[xc] + t - 1;
    xr = xrow[xp] - 1;
    for (yr = 0; yr < my; yr++) {
      yp = yr + xc * my;
      wp = yr + xr * my;
      dw[wp] += dy[yp] * xval[xp];
    }
    t += blockDim.x * gridDim.x;
  }
}

extern "C" {

  void A_mul_Bs_32(int mx, int ns,  float *x,  float *sval, int *srow, int *scol,  float *k) KCALL(_A_mul_Bs_32,mx,ns,x,sval,srow,scol,k);
  void A_mul_Bs_64(int mx, int ns, double *x, double *sval, int *srow, int *scol, double *k) KCALL(_A_mul_Bs_64,mx,ns,x,sval,srow,scol,k);
  void Ast_mul_Bs_32(int nx, int ns,  float *xval, int *xrow, int *xcol,  float *sval, int *srow, int *scol,  float *k) KCALL(_Ast_mul_Bs_32,nx,ns,xval,xrow,xcol,sval,srow,scol,k);
  void Ast_mul_Bs_64(int nx, int ns, double *xval, int *xrow, int *xcol, double *sval, int *srow, int *scol, double *k) KCALL(_Ast_mul_Bs_64,nx,ns,xval,xrow,xcol,sval,srow,scol,k);
  void As_mul_Bs_32(int mx, int ns,  float *xval, int *xrow, int *xcol,  float *sval, int *srow, int *scol,  float *k) KCALL(_As_mul_Bs_32,mx,ns,xval,xrow,xcol,sval,srow,scol,k);
  void As_mul_Bs_64(int mx, int ns, double *xval, int *xrow, int *xcol, double *sval, int *srow, int *scol, double *k) KCALL(_As_mul_Bs_64,mx,ns,xval,xrow,xcol,sval,srow,scol,k);

  void A_mul_Bst_32(int my, int ny, int mx, float *dy, float *xval, int *xrow, int *xcol, float *dw) {
    // dy[my,ny] * x[mx,nx]' -> w[mw,nw]   where ny=nx, mw=my, nw=mx
    CUDA(hipMemset(dw, 0, my * mx * sizeof(float)));
    CUDA(hipDeviceSynchronize());
    for (int xc=0; xc<ny; xc++) {		// do one column of x at a time (row of x')
      KCALL(_A_mul_Bst_32,my,xc,dy,xval,xrow,xcol,dw);
      CUDA(hipDeviceSynchronize());
    }
  }

  void A_mul_Bst_64(int my, int ny, int mx, double *dy, double *xval, int *xrow, int *xcol, double *dw) {
    // dy[my,ny] * x[mx,nx]' -> w[mw,nw]   where ny=nx, mw=my, nw=mx
    CUDA(hipMemset(dw, 0, my * mx * sizeof(double)));
    CUDA(hipDeviceSynchronize());
    for (int xc=0; xc<ny; xc++) {		// do one column of x at a time (row of x')
      KCALL(_A_mul_Bst_64,my,xc,dy,xval,xrow,xcol,dw);
      CUDA(hipDeviceSynchronize());
    }
  }

  // To test the blk,thr parameters:
  // #define KCALL(f,...) {f<<<BLK,THR>>>(__VA_ARGS__); CUDA(hipGetLastError()); }
  void At_test(int blk,int thr,int nx, int ns,  float *xval, int *xrow, int *xcol,  float *sval, int *srow, int *scol, float *k) {_Ast_mul_Bs_32<<<blk,thr>>>(nx,ns,xval,xrow,xcol,sval,srow,scol,k); CUDA(hipGetLastError()); }
  void A_test(int blk,int thr,int nx, int ns, float *xval, int *xrow, int *xcol, float *sval, int *srow, int *scol, float *k) {_As_mul_Bs_32<<<blk,thr>>>(nx,ns,xval,xrow,xcol,sval,srow,scol,k); CUDA(hipGetLastError()); }


}

template<typename dType>
__global__ void _log(int n, dType *a, dType *b) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    b[i] = log(a[i]);
    i += blockDim.x * gridDim.x;
  }
}

extern "C" {
  void log32(int n, float  *a, float  *b) KCALL(_log,n,a,b);
  void log64(int n, double *a, double *b) KCALL(_log,n,a,b);
}


template<typename dType>
__global__ void _exp(int n, dType *a, dType *b) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    b[i] = exp(a[i]);
    i += blockDim.x * gridDim.x;
  }
}

extern "C" {
  void exp32(int n, float  *a, float  *b) KCALL(_exp,n,a,b);
  void exp64(int n, double *a, double *b) KCALL(_exp,n,a,b);
}

template<typename dType>
__global__ void _diag(int m, int n, dType *a, dType *d) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int k = (m<n) ? m : n;
  while (i < k) {
    d[i] = a[i*m+i];
    i += blockDim.x * gridDim.x;
  }
}

extern "C" {
  void diag32(int m, int n, float  *a, float  *d) KCALL(_diag,m,n,a,d);
  void diag64(int m, int n, double *a, double *d) KCALL(_diag,m,n,a,d);
}

template<typename dType>
__global__ void _diagm(int m, int n, dType *d, dType *a) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  int k = (m<n) ? m : n;
  while (i < k) {
    a[i*m+i] = d[i];
    i += blockDim.x * gridDim.x;
  }
}

extern "C" {
  void diagm32(int m, int n, float  *d, float  *a) KCALL(_diagm,m,n,d,a);
  void diagm64(int m, int n, double *d, double *a) KCALL(_diagm,m,n,d,a);
}

