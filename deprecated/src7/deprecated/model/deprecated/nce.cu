#include "hip/hip_runtime.h"
#include "../knet.h"

template<typename dType>
__global__ void _nce_grad_real(int n, dType *ypred, dType *kqvec, dType *ygrad) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while(i < n) {
    int ij = n*i+i;
    ygrad[ij] = -(kqvec[i]/(exp(ypred[ij]) + kqvec[i]))/n;
    i += blockDim.x * gridDim.x;
  }
}

extern "C" {
  void nce_grad_real_32(int n, float  *ypred, float  *kqvec, float  *ygrad) KCALL(_nce_grad_real,n,ypred,kqvec,ygrad);
  void nce_grad_real_64(int n, double *ypred, double *kqvec, double *ygrad) KCALL(_nce_grad_real,n,ypred,kqvec,ygrad);
}

template<typename dType>
__global__ void _nce_loss_real(int n, dType *ypred, dType *kqvec, dType *ytemp) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while(i < n) {
    dType s = ypred[n*i+i];
    ytemp[i] = log(exp(s) + kqvec[i]) - s;
    i += blockDim.x * gridDim.x;
  }
}

extern "C" {
  void nce_loss_real_32(int n, float  *ypred, float  *kqvec, float  *ytemp) KCALL(_nce_loss_real,n,ypred,kqvec,ytemp);
  void nce_loss_real_64(int n, double *ypred, double *kqvec, double *ytemp) KCALL(_nce_loss_real,n,ypred,kqvec,ytemp);
}

template<typename dType>
__global__ void _nce_loss_noise(int K, int B, dType *ypred, dType *kqvec, dType *ytemp) {
  int kb = threadIdx.x + blockIdx.x * blockDim.x;
  int KB = K*B;
  while(kb < KB) {
    dType s = ypred[kb];
    dType kq = kqvec[kb % K];
    ytemp[kb] = -log(kq)+log(exp(s)+kq);
    kb += blockDim.x * gridDim.x;
  }
}

extern "C" {
  void nce_loss_noise_32(int K, int B, float  *ypred, float  *kqvec, float  *ytemp) KCALL(_nce_loss_noise,K,B,ypred,kqvec,ytemp);
  void nce_loss_noise_64(int K, int B, double *ypred, double *kqvec, double *ytemp) KCALL(_nce_loss_noise,K,B,ypred,kqvec,ytemp);
}

template<typename dType>
__global__ void _nce_grad_noise(int K, int B, dType *ypred, dType *kqvec, dType *ygrad) {
  int kb = threadIdx.x + blockIdx.x * blockDim.x;
  int KB = K*B;
  while(kb < KB) {
    dType exps = exp(ypred[kb]);
    dType kq = kqvec[kb % K];
    ygrad[kb] = (exps/(exps+kq))/B;
    kb += blockDim.x * gridDim.x;
  }
}

extern "C" {
  void nce_grad_noise_32(int K, int B, float  *ypred, float  *kqvec, float  *ygrad) KCALL(_nce_grad_noise,K,B,ypred,kqvec,ygrad);
  void nce_grad_noise_64(int K, int B, double *ypred, double *kqvec, double *ygrad) KCALL(_nce_grad_noise,K,B,ypred,kqvec,ygrad);
}

