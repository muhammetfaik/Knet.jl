#include "hip/hip_runtime.h"
#include "kunet.h"

/* kgauss uses the same algorithm and input/output format as At_mul_B
   to compute the gaussian kernel: x(nd,nx) s(nd,ns) -> k(nx,ns) 
   Input: two sparse matrices.  Output: a dense matrix.
   The sparse matrices are in 1-based csc format.
*/

__global__ void _kgauss32(int nx, int ns, float *xval, int *xrow, int *xcol, float *sval, int *srow, int *scol, float *k, float g) {
  int i, n, x1, x2, xc, xr, s1, s2, sc, sr;
  double d, dd;
  i = threadIdx.x + blockIdx.x * blockDim.x;
  n = nx*ns;
  while (i < n) {
    xc = i % nx;
    sc = i / nx;
    x1 = xcol[xc]-1; x2 = xcol[xc+1]-1;
    s1 = scol[sc]-1; s2 = scol[sc+1]-1;
    dd = 0;
    while ((x1 < x2) || (s1 < s2)) {
      xr = ((x1 < x2) ? xrow[x1] : INT_MAX);
      sr = ((s1 < s2) ? srow[s1] : INT_MAX);
      d = ((sr < xr) ? sval[s1++] :
	   (xr < sr) ? xval[x1++] :
	   (xval[x1++]-sval[s1++])); 
      dd += d*d;
    }
    k[i] = exp(-g * dd);
    i += blockDim.x * gridDim.x;
  }
}

__global__ void _kgauss64(int nx, int ns, double *xval, int *xrow, int *xcol, double *sval, int *srow, int *scol, double *k, double g) {
  int i, n, x1, x2, xc, xr, s1, s2, sc, sr;
  double d, dd;
  i = threadIdx.x + blockIdx.x * blockDim.x;
  n = nx*ns;
  while (i < n) {
    xc = i % nx;
    sc = i / nx;
    x1 = xcol[xc]-1; x2 = xcol[xc+1]-1;
    s1 = scol[sc]-1; s2 = scol[sc+1]-1;
    dd = 0;
    while ((x1 < x2) || (s1 < s2)) {
      xr = ((x1 < x2) ? xrow[x1] : INT_MAX);
      sr = ((s1 < s2) ? srow[s1] : INT_MAX);
      d = ((sr < xr) ? sval[s1++] :
	   (xr < sr) ? xval[x1++] :
	   (xval[x1++]-sval[s1++])); 
      dd += d*d;
    }
    k[i] = exp(-g * dd);
    i += blockDim.x * gridDim.x;
  }
}

/* kpoly uses the same algorithm and input/output format as At_mul_B
   to compute the polynomial kernel: x(nd,nx) s(nd,ns) -> k(nx,ns) */

__global__ void _kpoly32(int nx, int ns, float *xval, int *xrow, int *xcol, float *sval, int *srow, int *scol, float *k, float c, float d) {
  int i, n, x1, x2, xc, xr, s1, s2, sc, sr;
  i = threadIdx.x + blockIdx.x * blockDim.x;
  n = nx*ns;
  while (i < n) {
    double ki = 0;
    xc = i % nx;
    sc = i / nx;
    x1 = xcol[xc]-1; x2 = xcol[xc+1]-1;
    s1 = scol[sc]-1; s2 = scol[sc+1]-1;
    while ((x1 < x2) && (s1 < s2)) {
      xr = xrow[x1]; sr = srow[s1];
      if (sr < xr) s1++;
      else if (xr < sr) x1++;
      else ki += xval[x1++]*sval[s1++];
    }
    k[i] = pow(ki + c, (double) d);
    i += blockDim.x * gridDim.x;
  }
}

__global__ void _kpoly64(int nx, int ns, double *xval, int *xrow, int *xcol, double *sval, int *srow, int *scol, double *k, double c, double d) {
  int i, n, x1, x2, xc, xr, s1, s2, sc, sr;
  i = threadIdx.x + blockIdx.x * blockDim.x;
  n = nx*ns;
  while (i < n) {
    double ki = 0;
    xc = i % nx;
    sc = i / nx;
    x1 = xcol[xc]-1; x2 = xcol[xc+1]-1;
    s1 = scol[sc]-1; s2 = scol[sc+1]-1;
    while ((x1 < x2) && (s1 < s2)) {
      xr = xrow[x1]; sr = srow[s1];
      if (sr < xr) s1++;
      else if (xr < sr) x1++;
      else ki += xval[x1++]*sval[s1++];
    }
    k[i] = pow(ki + c, d);
    i += blockDim.x * gridDim.x;
  }
}

__global__ void _kpolymap32(int n, float *k, float c, float d) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    k[i] = pow(k[i] + c, d);
    i += blockDim.x * gridDim.x;
  }  
}

__global__ void _kpolymap64(int n, double *k, double c, double d) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    k[i] = pow(k[i] + c, d);
    i += blockDim.x * gridDim.x;
  }  
}

__global__ void _kgauss32map(int nx, int ns, float *x2, float *s2, float *k, float g) {
  int i, n, xi, si;
  i = threadIdx.x + blockIdx.x * blockDim.x;
  n = nx*ns;
  while (i < n) {
    xi = (i % nx);
    si = (i / nx);
    k[i] = exp(-g * (x2[xi] + s2[si] - 2*k[i]));
    i += blockDim.x * gridDim.x;
  }
}

__global__ void _kgauss64map(int nx, int ns, double *x2, double *s2, double *k, double g) {
  int i, n, xi, si;
  i = threadIdx.x + blockIdx.x * blockDim.x;
  n = nx*ns;
  while (i < n) {
    xi = (i % nx);
    si = (i / nx);
    k[i] = exp(-g * (x2[xi] + s2[si] - 2*k[i]));
    i += blockDim.x * gridDim.x;
  }
}

__global__ void _kgauss32sum(int xrows, int xcols, float *x, float *xx) {
  int i, j, x0, x1;
  double sum;
  j = threadIdx.x + blockIdx.x * blockDim.x;
  while (j < xcols) {
    x0 = j*xrows; x1 = x0+xrows;
    sum = 0;
    for (i=x0; i<x1; i++) sum += x[i]*x[i];
    xx[j] = sum;
    j += blockDim.x * gridDim.x;
  }
}

__global__ void _kgauss64sum(int xrows, int xcols, double *x, double *xx) {
  int i, j, x0, x1;
  double sum;
  j = threadIdx.x + blockIdx.x * blockDim.x;
  while (j < xcols) {
    x0 = j*xrows; x1 = x0+xrows;
    sum = 0;
    for (i=x0; i<x1; i++) sum += x[i]*x[i];
    xx[j] = sum;
    j += blockDim.x * gridDim.x;
  }
}

extern "C" {

  void kgauss32map(int nx, int ns, float *x2, float *s2, float *k, float g) KCALL(_kgauss32map,nx,ns,x2,s2,k,g);
  void kgauss32sum(int xrows, int xcols, float *x, float *x2) KCALL(_kgauss32sum,xrows,xcols,x,x2);

  void kgauss64map(int nx, int ns, double *x2, double *s2, double *k, double g) KCALL(_kgauss64map,nx,ns,x2,s2,k,g);
  void kgauss64sum(int xrows, int xcols, double *x, double *x2) KCALL(_kgauss64sum,xrows,xcols,x,x2);

  void kpolymap32(int n, float *k, float c, float d) KCALL(_kpolymap32,n,k,c,d);
  void kpolymap64(int n, double *k, double c, double d) KCALL(_kpolymap64,n,k,c,d);

  void kpoly32(int nx, int ns, float *xval, int *xrow, int *xcol, float *sval, int *srow, int *scol, float *k, float c, float d) KCALL(_kpoly32,nx,ns,xval,xrow,xcol,sval,srow,scol,k,c,d);
  void kpoly64(int nx, int ns, double *xval, int *xrow, int *xcol, double *sval, int *srow, int *scol, double *k, double c, double d) KCALL(_kpoly64,nx,ns,xval,xrow,xcol,sval,srow,scol,k,c,d);

  void kgauss32(int nx, int ns, float *xval, int *xrow, int *xcol, float *sval, int *srow, int *scol, float *k, float g) KCALL(_kgauss32,nx,ns,xval,xrow,xcol,sval,srow,scol,k,g);
  void kgauss64(int nx, int ns, double *xval, int *xrow, int *xcol, double *sval, int *srow, int *scol, double *k, double g) KCALL(_kgauss64,nx,ns,xval,xrow,xcol,sval,srow,scol,k,g);

}


/* DEAD CODE...
// no need for kback on gpu?

__global__ void _kback32(int nc, int nx, float *z, float *y, float *dw0, float *dw1, int *dj, float u) {
  int i0, i1, cz, cy;
  float cmax, ymax;
  int ix = threadIdx.x + blockIdx.x * blockDim.x;
  while (ix < nx) {
    i0 = ix * nc; 
    i1 = i0 + nc;
    ymax = -INFINITY; cy = -1;
    zmax = -INFINITY; cz = -1;
    for (int i=i0; i<i1; i++) {
      if (y[i] > ymax) { ymax = y[i]; cy = i; }
      if (z[i] > zmax) { zmax = z[i]; cz = i; }
    }
    if (cy != cz) {
      
    }
    ix += blockDim.x * gridDim.x;
  }
}


__global__ void _drop32(int n, float *x, float *xmask, float dropout, float scale) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    if (xmask[i] < dropout) x[i] = 0;
    else x[i] *= scale;
    i += blockDim.x * gridDim.x;
  }
}

__global__ void _drop64(int n, double *x, double *xmask, double dropout, double scale) {
  int i = threadIdx.x + blockIdx.x * blockDim.x;
  while (i < n) {
    if (xmask[i] < dropout) x[i] = 0;
    else x[i] *= scale;
    i += blockDim.x * gridDim.x;
  }
}

extern "C" {

void kback32(int nc, int nx, float *z, float *y, float *dw0, float *dw1, int *dj, int *dn, float *du) {

}

void drop32(int n, float *x, float *xmask, float dropout, float scale) KCALL(_drop32,n,x,xmask,dropout,scale);
void drop64(int n, double *x, double *xmask, double dropout, double scale) KCALL(_drop64,n,x,xmask,dropout,scale);



}

if GPU

function kback(l::KPerceptron, z::AbstractCudaArray{Float32})
    nptr = Cint[l.dn]; uptr = Cfloat[l.du]
    ccall((:kback32,libkunet), Void,
          (Ptr{Cfloat},Ptr{Cfloat},Ptr{Cfloat},Ptr{Cfloat},Ptr{Cint},Ptr{Cint},Ptr{Cfloat}),
          z,l.y,l.dw0,l.dw1,l.dj,nptr,uptr)
    l.dn = nptr[1]; l.du = uptr[1]
end

function kback(l::KPerceptron, z::AbstractCudaArray{Float64})
    nptr = Cint[l.dn]; uptr = Cdouble[l.du]
    ccall((:kback64,libkunet), Void,
          (Ptr{Cdouble},Ptr{Cdouble},Ptr{Cdouble},Ptr{Cdouble},Ptr{Cint},Ptr{Cint},Ptr{Cdouble}),
          z,l.y,l.dw0,l.dw1,l.dj,nptr,uptr)
    l.dn = nptr[1]; l.du = uptr[1]
end

end # if GPU

// buggy: does not process when one matrix has a zero 
__global__ void _kgauss32(int mx, int ns, float *xval, int *xrow, int *xcol, float *sval, int *srow, int *scol, float g, float *k) {
  // assume x(mx,nd) and s(nd,ns) are in 1-based csc format
  // assume k(mx,ns) has been allocated and zeroed out
  int s0, s1, sp, sc, sr, x0, x1, xp, xc, xr, k0, k1, kp;
  float sv, xv, xs;
  sc = threadIdx.x + blockIdx.x * blockDim.x;
  k0 = mx*sc;		// k[k0]: first element of k[:,sc]
  k1 = k0+mx;		// k[k1-1]: last element of k[:,sc]
  while (sc < ns) {	// sc: 0-based column for s
    s0 = scol[sc]-1;    // first element of s[:,sc] is at sval[s0] (scol entries are 1-based)
    s1 = scol[sc+1]-1;  // last element of s[:,sc] is at sval[s1-1]
    for (sp = s0; sp < s1; sp++) {
      sr = srow[sp]-1;  // sr: 0-based row for s (srow entries are 1-based)
      sv = sval[sp];	// sv: s[sr,sc] (0-based)
      xc = sr;		// xc: 0-based column for x (=sr)
      x0 = xcol[xc]-1;  // first element of x[:,xc] is at xval[x0]
      x1 = xcol[xc+1]-1; // last element of x[:,xc] is at xval[x1-1]
      for (xp = x0; xp < x1; xp++) {
	xr = xrow[xp]-1; // xr: 0-based row for x
	xv = xval[xp];	 // xv: x[xr,xc=sr], now we can set k[xr,sc]
	xs = xv - sv;
	k[k0+xr] += xs*xs; // k += (xi-si)^2
      }
    }
    for (kp = k0; kp < k1; kp++) {
      k[kp] = exp(-g*k[kp]); // k = exp(-g*sum((xi-si)^2))
    }
    sc += blockDim.x * gridDim.x;
  }
}

extern "C" {
  void kgauss32(int mx, int ns, float *xval, int *xrow, int *xcol, float *sval, int *srow, int *scol, float g, float *k) {
    float *x2 = calloc(mx, sizeof(float));
    float *s2 = calloc(ns, sizeof(float));
    KCALL(_rowsq,mx,xval,xrow,xcol,x2);
    KCALL(_colsq,ns,sval,srow,scol,s2);
    KCALL(_kgauss32,mx,ns,xval,xrow,xcol,sval,srow,scol,g,k)
  }
}

__global__ void _kgauss32d(int nx, int ns, int nd, float *x, float *s, float *k, float g) {
  int i, j, n, xj, sj;
  double d, dd;
  i = threadIdx.x + blockIdx.x * blockDim.x;
  n = nx*ns;
  while (i < n) {
    xj = (i % nx)*nd;
    sj = (i / nx)*nd;
    dd = 0;
    for (j = 0; j < nd; j++) {
      d = x[xj++]-s[sj++];
      dd += d*d;
    }
    k[i] = exp(-g * dd);
    i += blockDim.x * gridDim.x;
  }
}

__global__ void _kgauss64d(int nx, int ns, int nd, double *x, double *s, double *k, double g) {
  int i, j, n, xj, sj;
  double d, dd;
  i = threadIdx.x + blockIdx.x * blockDim.x;
  n = nx*ns;
  while (i < n) {
    xj = (i % nx)*nd;
    sj = (i / nx)*nd;
    dd = 0;
    for (j = 0; j < nd; j++) {
      d = x[xj++]-s[sj++];
      dd += d*d;
    }
    k[i] = exp(-g * dd);
    i += blockDim.x * gridDim.x;
  }
}

  void kgauss32d(int nx, int ns, int nd, float *x, float *s, float *k, float g) KCALL(_kgauss32d,nx,ns,nd,x,s,k,g);
  void kgauss64d(int nx, int ns, int nd, double *x, double *s, double *k, double g) KCALL(_kgauss64d,nx,ns,nd,x,s,k,g);

*/
